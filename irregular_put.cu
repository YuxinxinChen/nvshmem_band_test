#include "hip/hip_runtime.h"
#include <iostream>

#include <hip/hip_runtime.h>

#include <shmem.h>

#include "time.cuh"


#define TID (threadIdx.x+blockIdx.x*blockDim.x)
#define WARPID ((threadIdx.x+blockIdx.x*blockDim.x)>>5)


__global__ void full_band(size_t size, int *remote_buffer, int *local_buffer, int remote_pe)
{
  for(uint32_t i = TID; i<size; i+=blockDim.x*gridDim.x)
  {
//    int item = shmem_int_g(remote_buffer+i, remote_pe);
//    local_buffer[i] = item;
    shmem_int_put(remote_buffer+i, local_buffer+i, 1, remote_pe);
  }
}

__global__ void char_band(size_t size, int *remote_buffer, int *local_buffer, int remote_pe)
{
  for(uint32_t i=TID; i<size; i+=blockDim.x*gridDim.x)
  {
    shmem_putmem((void *)(remote_buffer+i), (void *)(local_buffer+i), sizeof(int), remote_pe);
  }
}



int main()
{
  size_t size = 1<<20;
  shmem_init();
  int my_pe = shmem_my_pe();
  int n_pe = shmem_n_pes();

  int dev_count;
  hipGetDeviceCount(&dev_count);
  hipSetDevice(my_pe);

  int * remote_buffer = (int *)shmem_malloc(sizeof(int)*size);
  int * local_buffer;
  hipMallocManaged(&local_buffer, sizeof(int)*size);
  int remote_pe = my_pe^1;

  GpuTimer timer;
  float totaltime = 0.0;

  for(int i=0; i<200; i++)
  {
    timer.Start();
    char_band<<<320, 512>>>(size, remote_buffer, local_buffer, remote_pe);
    hipDeviceSynchronize();
    timer.Stop();
    totaltime = totaltime + timer.ElapsedMillis();
  }

  shmem_barrier_all();
  shmem_finalize();
  totaltime = totaltime/200;
  std::cout <<"PE "<< my_pe <<  " average time: " <<  totaltime << " bandwithd: "<<(sizeof(int)*size/(totaltime/1000))/(1024*1024*1024)<<" GB/s" << std::endl;
  std::cout << "end of the program: "<< my_pe << std::endl;
  return 0;
}
