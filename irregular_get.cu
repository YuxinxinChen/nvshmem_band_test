#include "hip/hip_runtime.h"
#include <iostream>

#include <hip/hip_runtime.h>

#include <shmem.h>
#include <shmemx.h>

#include "time.cuh"


#define TID (threadIdx.x+blockIdx.x*blockDim.x)
#define WARPID ((threadIdx.x+blockIdx.x*blockDim.x)>>5)


__global__ void full_band(size_t size, int *remote_buffer, int *local_buffer, int remote_pe)
{
  for(uint32_t i = TID; i<size; i+=blockDim.x*gridDim.x)
  {
    shmem_int_get(local_buffer+i, remote_buffer+i, 1, remote_pe);
  }
}
__global__ void full_band_warp(size_t size, int *remote_buffer, int *local_buffer, int remote_pe)
{
  for(uint32_t i=TID; i<size; i+=blockDim.x*gridDim.x)
  {
    i = __shfl_sync(0xffffffff, i, 0);
    __syncwarp();
    shmemx_int_get_warp(local_buffer+i, remote_buffer+i, 32, remote_pe);
  }
}
__global__ void full_band_block(size_t size, int *remote_buffer, int *local_buffer, int remote_pe)
{
  for(uint32_t i=TID; i<size; i+=blockDim.x*gridDim.x)
  {
    shmemx_int_get_block(local_buffer+blockIdx.x*blockDim.x, remote_buffer+blockIdx.x*blockDim.x, blockDim.x, remote_pe);
  }
}
__global__ void char_band(size_t size, int *remote_buffer, int *local_buffer, int remote_pe)
{
  for(uint32_t i=TID; i<size; i+=blockDim.x*gridDim.x)
  {
    shmem_getmem((void *)(local_buffer+i), (void *)(remote_buffer+i), sizeof(int), remote_pe);
  }
}



int main()
{
  size_t size = 1<<25;
  shmem_init();
  int my_pe = shmem_my_pe();
  int n_pes = shmem_n_pes();

  int dev_count;
  hipGetDeviceCount(&dev_count);
  hipSetDevice(my_pe);

  int * remote_buffer = (int *)shmem_malloc(sizeof(int)*size*2);
  int * local_buffer;
  hipMallocManaged(&local_buffer, sizeof(int)*size*2);

  GpuTimer timer;
  float totaltime = 0.0;
  int num_round = 200;
  hipStream_t *streams;
  streams = (hipStream_t *)malloc(sizeof(hipStream_t)*(n_pes-1));
  for(int i = 0; i<n_pes-1; i++ )
      hipStreamCreateWithFlags(streams+i, hipStreamNonBlocking);
  shmem_barrier_all();

  for(int i=0; i<num_round; i++)
  {
    int remote_pe = (my_pe+1)%n_pes;
    timer.Start();
    for(int j=0; j<n_pes-1; j++)
    {
    char_band<<<80, 512, 0, streams[j]>>>(size, remote_buffer, local_buffer, remote_pe);
    remote_pe = (remote_pe+1) % n_pes;
    }
    hipDeviceSynchronize();
    timer.Stop();
    totaltime = totaltime + timer.ElapsedMillis();
  }

  shmem_barrier_all();
  shmem_finalize();
  totaltime = totaltime/num_round;
  std::cout <<"PE "<< my_pe <<  " average time: " <<  totaltime << " bandwithd: "<<(sizeof(int)*size*(n_pes-1)/(totaltime/1000))/(1024*1024*1024)<<" GB/s" << std::endl;
  std::cout << "end of the program: "<< my_pe << std::endl;
  return 0;
}
